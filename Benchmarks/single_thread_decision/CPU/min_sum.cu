#include "hip/hip_runtime.h"
/*===================================================================================
	MIN_SUM.C
=====================================================================================

Contains:
  -> Procedures for simulating MIN-SUM algorithms

Written by: Oscar Ferraz from the code of Marco Alexandre Cravo Gomes 
			University of Coimbra - Electrical Engineering and Computer Department
			
			email: <oscar.ferraz@co.it.pt>
			
Date: March 2023
=====================================================================================*/



//==============================================//
// I N C L U D E S								//
//==============================================//
//---------------------- ANSI C
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <errno.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>  

#define N 128
#define M 64
#define EDGES 512
#define MAX_ITER 5



__constant__ unsigned char d_LUT_VN[EDGES];
__constant__ unsigned char d_LUT_CN[EDGES];

__global__ void GPU_min_sum_sindrome(unsigned short * d_sindrome, unsigned char * d_LPi);
__global__ void GPU_min_sum_equations_and_decision(unsigned short * d_sindrome, unsigned char * d_LPi, unsigned char * d_wrong_equations, unsigned char * d_teta, unsigned char * d_bar);


struct timespec start, end;	



//**************************************************************************************************
//Kernel
__global__ void GPU_min_sum_sindrome(unsigned short * d_sindrome, unsigned char * d_LPi){

    //unsigned char x=threadIdx.x+blockIdx.x*blockDim.x;
    
	short k;
       
    k=threadIdx.x*8;
    unsigned short sum=0;
    for(int j=0; j<8;j++){ 
        sum=sum+d_LPi[d_LUT_CN[k+j]];
    }
    d_sindrome[threadIdx.x]=sum & 0x1;
    //printf("sum=%d\n", sum);
    //printf("sindrome[%d]=%d\n", i, sindrome[i]);  
        

} 

//**************************************************************************************************
//Kernel
__global__ void GPU_min_sum_equations_and_decision(unsigned short * d_sindrome, unsigned char * d_LPi, unsigned char * d_wrong_equations, unsigned char * d_teta, unsigned char * d_bar){

    //unsigned char x=threadIdx.x+blockIdx.x*blockDim.x;
    
	short k;
        
    if(threadIdx.x<64){
        k=threadIdx.x*5;
        
        unsigned short sum=0;
        for(int j=0; j<5;j++){ 
            sum=sum+d_sindrome[d_LUT_VN[k+j]];
        }
        d_wrong_equations[threadIdx.x]=sum;
    }

    if(threadIdx.x>63){
        k=5*63+threadIdx.x*3;

        unsigned short sum=0;
        for(int j=0; j<3;j++){ 
            sum=sum+d_sindrome[d_LUT_VN[k+j]];
        }
        d_wrong_equations[threadIdx.x]=sum;
    }
    __syncthreads();

    
    if(d_wrong_equations[threadIdx.x]>=d_bar[0]){
        if(d_LPi[threadIdx.x]==0){
            d_LPi[threadIdx.x]=1;
        }
        else{
            d_LPi[threadIdx.x]=0;
        }
        d_teta[0]=0;
        
    }
    //printf("sindrome[%d]=%d\n", i, L[i]);

} 


/*************************************************************************************************************/
/*																											 */
/*  M A I N   P R O G R A M																					 */		
/*																											 */
/*************************************************************************************************************/
int  main(){

    unsigned char LUT_VN[EDGES]={0,9,26,44,48,1,10,27,45,49,2,11,28,46,50,3,12,29,47,51,4,13,30,32,52,5,14,31,33,53,6,15,16,34,54,0,7,17,35,55,1,8,18,36,56,2,9,19,37,57,3,10,20,38,58,4,11,21,39,59,5,12,22,40,60,6,13,23,41,61,7,14,24,42,62,8,15,25,43,63,14,16,17,47,63,15,17,18,32,48,0,18,19,33,49,1,19,20,34,50,2,20,21,35,51,3,21,22,36,52,4,22,23,37,53,5,23,24,38,54,6,24,25,39,55,7,25,26,40,56,8,26,27,41,57,9,27,28,42,58,10,28,29,43,59,11,29,30,44,60,12,30,31,45,61,13,16,31,46,62,2,16,32,33,55,3,17,33,34,56,4,18,34,35,57,5,19,35,36,58,6,20,36,37,59,7,21,37,38,60,8,22,38,39,61,9,23,39,40,62,10,24,40,41,63,11,25,41,42,48,12,26,42,43,49,13,27,43,44,50,14,28,44,45,51,15,29,45,46,52,0,30,46,47,53,1,31,32,47,54,10,31,34,48,51,11,16,35,49,52,12,17,36,50,53,13,18,37,51,54,14,19,38,52,55,15,20,39,53,56,0,21,40,54,57,1,22,41,55,58,2,23,42,56,59,3,24,43,57,60,4,25,44,58,61,5,26,45,59,62,6,27,46,60,63,7,28,47,48,61,8,29,32,49,62,9,30,33,50,63,16,37,50,17,38,51,18,39,52,19,40,53,20,41,54,21,42,55,22,43,56,23,44,57,24,45,58,25,46,59,26,47,60,27,32,61,28,33,62,29,34,63,30,35,48,31,36,49,0,32,63,1,33,48,2,34,49,3,35,50,4,36,51,5,37,52,6,38,53,7,39,54,8,40,55,9,41,56,10,42,57,11,43,58,12,44,59,13,45,60,14,46,61,15,47,62,3,16,48,4,17,49,5,18,50,6,19,51,7,20,52,8,21,53,9,22,54,10,23,55,11,24,56,12,25,57,13,26,58,14,27,59,15,28,60,0,29,61,1,30,62,2,31,63,0,25,45,1,26,46,2,27,47,3,28,32,4,29,33,5,30,34,6,31,35,7,16,36,8,17,37,9,18,38,10,19,39,11,20,40,12,21,41,13,22,42,14,23,43,15,24,44};
    unsigned char LUT_CN[EDGES]={0,7,18,46,54,80,109,112,1,8,19,47,55,81,110,113,2,9,20,32,56,82,111,114,3,10,21,33,57,83,96,115,4,11,22,34,58,84,97,116,5,12,23,35,59,85,98,117,6,13,24,36,60,86,99,118,7,14,25,37,61,87,100,119,8,15,26,38,62,88,101,120,0,9,27,39,63,89,102,121,1,10,28,40,48,90,103,122,2,11,29,41,49,91,104,123,3,12,30,42,50,92,105,124,4,13,31,43,51,93,106,125,5,14,16,44,52,94,107,126,6,15,17,45,53,95,108,127,6,16,31,32,49,64,96,119,7,16,17,33,50,65,97,120,8,17,18,34,51,66,98,121,9,18,19,35,52,67,99,122,10,19,20,36,53,68,100,123,11,20,21,37,54,69,101,124,12,21,22,38,55,70,102,125,13,22,23,39,56,71,103,126,14,23,24,40,57,72,104,127,15,24,25,41,58,73,105,112,0,25,26,42,59,74,106,113,1,26,27,43,60,75,107,114,2,27,28,44,61,76,108,115,3,28,29,45,62,77,109,116,4,29,30,46,63,78,110,117,5,30,31,47,48,79,111,118,4,17,32,47,62,75,80,115,5,18,32,33,63,76,81,116,6,19,33,34,48,77,82,117,7,20,34,35,49,78,83,118,8,21,35,36,50,79,84,119,9,22,36,37,51,64,85,120,10,23,37,38,52,65,86,121,11,24,38,39,53,66,87,122,12,25,39,40,54,67,88,123,13,26,40,41,55,68,89,124,14,27,41,42,56,69,90,125,15,28,42,43,57,70,91,126,0,29,43,44,58,71,92,127,1,30,44,45,59,72,93,112,2,31,45,46,60,73,94,113,3,16,46,47,61,74,95,114,0,17,41,48,61,78,81,96,1,18,42,49,62,79,82,97,2,19,43,50,63,64,83,98,3,20,44,48,51,65,84,99,4,21,45,49,52,66,85,100,5,22,46,50,53,67,86,101,6,23,47,51,54,68,87,102,7,24,32,52,55,69,88,103,8,25,33,53,56,70,89,104,9,26,34,54,57,71,90,105,10,27,35,55,58,72,91,106,11,28,36,56,59,73,92,107,12,29,37,57,60,74,93,108,13,30,38,58,61,75,94,109,14,31,39,59,62,76,95,110,15,16,40,60,63,77,80,111};


    hipError_t err=hipSuccess; 

    //======================================================================================================================================================================
    //kernel dimensions
    dim3 numBlocks(1,1,1);
    dim3 threadsPerBlock_sindrome(64,1,1);
    dim3 threadsPerBlock_equations_and_decision(128,1,1);

    //======================================================================================================================================================================
    //size of variables

    /* size_t size_local_sum=sizeof(unsigned int)*input_params.x_size*input_params.y_size*input_params.z_size;
    size_t size_scaled_predicted=sizeof(signed int)*input_params.x_size*input_params.y_size*input_params.z_size;
    size_t size_omega=sizeof(unsigned short)*input_params.x_size*input_params.y_size*input_params.z_size;
    size_t size_samples=(sizeof(unsigned short int)*input_params.x_size*input_params.y_size*input_params.z_size);
    size_t size_sign_scaled=(sizeof(unsigned short int)*input_params.x_size*input_params.y_size*input_params.z_size); */
    size_t size_LPi=(sizeof(unsigned char)*N); 
    size_t size_LUT=(sizeof(unsigned char)*EDGES); 
    size_t size_sindrome=(sizeof(unsigned short)*M); 
    size_t size_char=(sizeof(unsigned char)); 

    //==================================================================================================================LPi====================================================
    //variables declaration

    unsigned char *h_LPi=NULL;
    unsigned char *h_decoded_word=NULL;
    unsigned short *d_sindrome=NULL;
    unsigned char *d_wrong_equations=NULL;
    unsigned char *d_LPi=NULL;
    unsigned char *h_bar=NULL;
    unsigned char *h_teta=NULL;
    unsigned char *d_bar=NULL;
    unsigned char *d_teta=NULL;


    //======================================================================================================================================================================
    //allocate host memory

    err=hipHostAlloc((void **)&h_LPi, size_LPi, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host LPi(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostAlloc((void **)&h_decoded_word, size_LPi, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host decoded_word(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostAlloc((void **)&h_bar, size_char, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host h_bar(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostAlloc((void **)&h_teta, size_char, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host decoded_word(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    //======================================================================================================================================================================
    //Initialize host memory
    h_teta[0]=0;
    h_bar[0]=5;


    {
        h_LPi[0]=1;
        h_LPi[1]=0;
        h_LPi[2]=1;
        h_LPi[3]=1;
        h_LPi[4]=0;
        h_LPi[5]=0;
        h_LPi[6]=1;
        h_LPi[7]=1;
        h_LPi[8]=0;
        h_LPi[9]=0;
        h_LPi[10]=1;
        h_LPi[11]=0;
        h_LPi[12]=0;
        h_LPi[13]=1;
        h_LPi[14]=1;
        h_LPi[15]=0;
        h_LPi[16]=0;
        h_LPi[17]=0;
        h_LPi[18]=1;
        h_LPi[19]=0;
        h_LPi[20]=0;
        h_LPi[21]=1;
        h_LPi[22]=0;
        h_LPi[23]=0;
        h_LPi[24]=0;
        h_LPi[25]=1;
        h_LPi[26]=1;
        h_LPi[27]=1;
        h_LPi[28]=0;
        h_LPi[29]=0;
        h_LPi[30]=0;
        h_LPi[31]=1;
        h_LPi[32]=0;
        h_LPi[33]=0;
        h_LPi[34]=0;
        h_LPi[35]=0;
        h_LPi[36]=0;
        h_LPi[37]=1;
        h_LPi[38]=0;
        h_LPi[39]=0;
        h_LPi[40]=1;
        h_LPi[41]=1;
        h_LPi[42]=0;
        h_LPi[43]=0;
        h_LPi[44]=0;
        h_LPi[45]=0;
        h_LPi[46]=0;
        h_LPi[47]=0;
        h_LPi[48]=1;
        h_LPi[49]=1;
        h_LPi[50]=1;
        h_LPi[51]=0;
        h_LPi[52]=1;
        h_LPi[53]=1;
        h_LPi[54]=1;
        h_LPi[55]=0;
        h_LPi[56]=1;
        h_LPi[57]=0;
        h_LPi[58]=0;
        h_LPi[59]=0;
        h_LPi[60]=0;
        h_LPi[61]=1;
        h_LPi[62]=1;
        h_LPi[63]=0;
        h_LPi[64]=0;
        h_LPi[65]=0;
        h_LPi[66]=0;
        h_LPi[67]=0;
        h_LPi[68]=0;
        h_LPi[69]=0;
        h_LPi[70]=0;
        h_LPi[71]=1;
        h_LPi[72]=0;
        h_LPi[73]=0;
        h_LPi[74]=1;
        h_LPi[75]=0;
        h_LPi[76]=0;
        h_LPi[77]=0;
        h_LPi[78]=1;
        h_LPi[79]=0;
        h_LPi[80]=0;
        h_LPi[81]=0;
        h_LPi[82]=1;
        h_LPi[83]=0;
        h_LPi[84]=0;
        h_LPi[85]=0;
        h_LPi[86]=0;
        h_LPi[87]=1;
        h_LPi[88]=1;
        h_LPi[89]=1;
        h_LPi[90]=1;
        h_LPi[91]=1;
        h_LPi[92]=1;
        h_LPi[93]=1;
        h_LPi[94]=0;
        h_LPi[95]=0;
        h_LPi[96]=0;
        h_LPi[97]=0;
        h_LPi[98]=0;
        h_LPi[99]=0;
        h_LPi[100]=0;
        h_LPi[101]=0;
        h_LPi[102]=0;
        h_LPi[103]=1;
        h_LPi[104]=0;
        h_LPi[105]=0;
        h_LPi[106]=0;
        h_LPi[107]=0;
        h_LPi[108]=0;
        h_LPi[109]=1;
        h_LPi[110]=0;
        h_LPi[111]=0;
        h_LPi[112]=1;
        h_LPi[113]=1;
        h_LPi[114]=1;
        h_LPi[115]=0;
        h_LPi[116]=0;
        h_LPi[117]=0;
        h_LPi[118]=1;
        h_LPi[119]=1;
        h_LPi[120]=0;
        h_LPi[121]=0;
        h_LPi[122]=0;
        h_LPi[123]=0;
        h_LPi[124]=1;
        h_LPi[125]=0;
        h_LPi[126]=0;
        h_LPi[127]=1;
    }

    //======================================================================================================================================================================
    //allocate device memory

    err=hipMalloc((void **)&d_sindrome, size_sindrome);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_sindrome (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMemset(d_sindrome, 0, size_sindrome);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to set device d_sindrome (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_LPi, size_LPi);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_LPi (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_wrong_equations, size_LPi);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_wrong_equations (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_teta, size_char);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device teta (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_bar, size_char);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device bar (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }



    //======================================================================================================================================================================
    //copy data to device

    clock_gettime(CLOCK_MONOTONIC, &start);
      
    err=hipMemcpy(d_LPi, h_LPi, size_LPi, hipMemcpyHostToDevice);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy the LPi from host to device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMemcpyToSymbol(HIP_SYMBOL(d_LUT_VN), &LUT_VN, size_LUT);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy LUT_VN from host to constant (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMemcpyToSymbol(HIP_SYMBOL(d_LUT_CN), &LUT_CN, size_LUT);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy LUT_CN from host to constant (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_bar, h_bar, size_char, hipMemcpyHostToDevice);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy bar from host to device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_teta, h_teta, size_char, hipMemcpyHostToDevice);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy teta from host to device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    //======================================================================================================================================================================
    //execute the kernel
    for(int num_iter=0; num_iter<MAX_ITER;num_iter++){

        err=hipMemcpy(h_teta , d_teta, size_char, hipMemcpyDeviceToHost);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to copy the d_teta from device to host (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        } 

        err=hipMemcpy(h_bar , d_bar, size_char, hipMemcpyDeviceToHost);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to copy the bar from device to host (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        } 

        if(h_teta[0] && h_bar[0] > 1){

            h_bar[0] = h_bar[0] - 1;
        }
        h_teta[0]=1;

        err=hipMemcpy(d_bar, h_bar, size_char, hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to copy bar from host to device (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }

        err=hipMemcpy(d_teta, h_teta, size_char, hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to copy teta from host to device (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }
    
        

        GPU_min_sum_sindrome<<<numBlocks, threadsPerBlock_sindrome>>>(d_sindrome, d_LPi);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to launch sindrome kernel (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }

        GPU_min_sum_equations_and_decision<<<numBlocks, threadsPerBlock_equations_and_decision>>>(d_sindrome, d_LPi, d_wrong_equations, d_teta, d_bar);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to launch equations_and_decision kernel (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }
    }
    //======================================================================================================================================================================
    //copy the data from device to host
    err=hipMemcpy(h_decoded_word , d_LPi, size_LPi, hipMemcpyDeviceToHost);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy the decoded_word from device to host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    } 

    clock_gettime(CLOCK_MONOTONIC, &end);
    

    //======================================================================================================================================================================
    //free the device memory

    err=hipFree(d_LPi);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_LPi from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_sindrome);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_sindrome from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_wrong_equations);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_wrong_equations from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_teta);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_teta from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_bar);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_bar from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }
    

    //======================================================================================================================================================================
    //Print Results
    printf("Execution time=%lfms\n",(end.tv_sec-start.tv_sec)*1e3+(end.tv_nsec-start.tv_nsec)*1e-6);

    for (int i=0; i<128 ;i++){
        printf("%d\n",  h_decoded_word[i] );
    } 


    //======================================================================================================================================================================
    //free the host memory

    err=hipHostFree(h_LPi);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_LPi from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostFree(h_decoded_word);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_decoded_word from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostFree(h_teta);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_teta from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostFree(h_bar);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_bar from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    
		
   return 0;


} 


